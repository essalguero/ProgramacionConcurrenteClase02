#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>

__global__ void kernel(int *a)
{
	a[blockIdx.x * blockDim.x + threadIdx.x ] = 0;
}

double tiempo( void )
{
	struct timeval  tv;
	gettimeofday(&tv, NULL);

	return (double) (tv.tv_usec) / 1000000 + (double) (tv.tv_sec);
}

int main(int argc, char** argv)
{
	double tiempoInicio;
	double tiempoFin;
	
	int n;
	if (argc == 2)
	{
		n = atoi(argv[1]);
	} else {
		n = 64;
	}

	printf("\nElementos a reservar: %d\n\n\n", n);

	int numBytes = n * sizeof(int);

	int *d_a;
	int *h_a;

	hipMalloc((void **) &d_a, numBytes );

	h_a = (int *)malloc(numBytes);

	dim3 blockSize(8);
	dim3 gridSize(8);


	tiempoInicio = tiempo();
	kernel <<<gridSize, blockSize>>>(d_a);
	hipDeviceSynchronize();
	tiempoFin = tiempo();
	
	if ( hipSuccess != hipGetLastError() )
		printf( "Error!\n" );

	printf("Tiempo de inicio Kernel: %lf\n", tiempoInicio);
	printf("Tiempo de fin Kernel: %lf\n", tiempoFin);
	printf("Tiempo total: %lf\n\n\n", tiempoFin - tiempoInicio);


	tiempoInicio = tiempo();
	hipMemcpy (d_a, h_a, numBytes, hipMemcpyDeviceToHost);
	tiempoFin = tiempo();

	printf("Tiempo de inicio Transferencia: %lf\n", tiempoInicio);
	printf("Tiempo de fin Transferencia: %lf\n", tiempoFin);
	printf("Tiempo total: %lf\n", tiempoFin - tiempoInicio);

	printf("Done.\n");

	return 0;
}
